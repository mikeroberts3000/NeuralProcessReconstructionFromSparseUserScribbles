#include "hip/hip_runtime.h"
#include "Mojo.Core/D3D11.hpp"
#include "Mojo.Core/Cuda.hpp"
#include "Mojo.Core/Thrust.hpp"
#include "Mojo.Core/SegmenterState.hpp"
#include "Mojo.Core/ID3D11CudaTexture.hpp"
#include "Mojo.Core/ForEach.hpp"

#include "Index.cuh"

extern "C" void InitializeCommittedSegmentation( Mojo::Core::SegmenterState* segmenterState )
{
    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< uchar4 >( "ColorMap" ).begin(),
            segmenterState->deviceVectors.Get< uchar4 >( "ColorMap" ).end(),
            segmenterState->parameters.Get< uchar4 >( "COLOR_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );

    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< int >( "IdMap" ).begin(),
            segmenterState->deviceVectors.Get< int >( "IdMap" ).end(),
            segmenterState->parameters.Get< int >( "ID_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );
}

extern "C" void InitializeSegmentation( Mojo::Core::SegmenterState* segmenterState )
{   
    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "PrimalMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "PrimalMap" ).end(),
            segmenterState->parameters.Get< float >( "PRIMAL_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );

    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "OldPrimalMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "OldPrimalMap" ).end(),
            segmenterState->parameters.Get< float >( "PRIMAL_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );

    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float4 >( "DualMap" ).begin(),
            segmenterState->deviceVectors.Get< float4 >( "DualMap" ).end(),
            segmenterState->parameters.Get< float4 >( "DUAL_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float4 >( "ScratchpadMap" ) ) );
}

extern "C" void InitializeConstraintMap( Mojo::Core::SegmenterState* segmenterState )
{
    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "ConstraintMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "ConstraintMap" ).end(),
            segmenterState->parameters.Get< float >( "CONSTRAINT_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );
}

extern "C" void InitializeScratchpad( Mojo::Core::SegmenterState* segmenterState )
{
    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ).end(),
            segmenterState->parameters.Get< float >( "SCRATCHPAD_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );
}

extern "C" void InitializeCostMap( Mojo::Core::SegmenterState* segmenterState )
{
    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "CostForwardMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "CostForwardMap" ).end(),
            segmenterState->parameters.Get< float >( "COST_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );

    MOJO_THRUST_SAFE(
        Mojo::Core::Thrust::Fill(
            segmenterState->deviceVectors.Get< float >( "CostBackwardMap" ).begin(),
            segmenterState->deviceVectors.Get< float >( "CostBackwardMap" ).end(),
            segmenterState->parameters.Get< float >( "COST_MAP_INITIAL_VALUE" ),
            segmenterState->deviceVectors.Get< float >( "ScratchpadMap" ) ) );
}